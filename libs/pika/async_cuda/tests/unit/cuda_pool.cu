#include "hip/hip_runtime.h"
//  Copyright (c) 2021 ETH Zurich
//
//  SPDX-License-Identifier: BSL-1.0
//  Distributed under the Boost Software License, Version 1.0. (See accompanying
//  file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <pika/async_cuda/cuda_exception.hpp>
#include <pika/async_cuda/cuda_pool.hpp>
#include <pika/modules/testing.hpp>

#include <cstddef>
#include <utility>
#include <vector>

namespace cu = pika::cuda::experimental;

__global__ void kernel(int* p, int i)
{
    p[i] = i * 2;
}

int main()
{
    {
        // A pool with only one stream always gives the same stream
        cu::cuda_pool pool{0, 1, 1};

        auto& stream1 = pool.get_next_stream();
        auto& stream2 = pool.get_next_stream();
        auto& stream3 = pool.get_next_stream();

        PIKA_TEST_EQ(stream1, stream2);
        PIKA_TEST_EQ(stream1, stream3);

        auto& hpstream1 =
            pool.get_next_stream(pika::threads::thread_priority::high);
        auto& hpstream2 =
            pool.get_next_stream(pika::threads::thread_priority::high);
        auto& hpstream3 =
            pool.get_next_stream(pika::threads::thread_priority::high);

        PIKA_TEST_EQ(hpstream1, hpstream2);
        PIKA_TEST_EQ(hpstream1, hpstream3);
    }

    {
        // A pool with multiple streams cycles through the streams
        cu::cuda_pool pool{0, 3, 2};

        auto& stream1 = pool.get_next_stream();
        auto& stream2 = pool.get_next_stream();
        auto& stream3 = pool.get_next_stream();
        auto& stream4 = pool.get_next_stream();
        auto& stream5 = pool.get_next_stream();
        auto& stream6 = pool.get_next_stream();

        PIKA_TEST_EQ(stream1, stream4);
        PIKA_TEST_EQ(stream2, stream5);
        PIKA_TEST_EQ(stream3, stream6);
        PIKA_TEST_NEQ(stream1, stream2);
        PIKA_TEST_NEQ(stream1, stream3);
        PIKA_TEST_NEQ(stream2, stream3);

        auto& hpstream1 =
            pool.get_next_stream(pika::threads::thread_priority::high);
        auto& hpstream2 =
            pool.get_next_stream(pika::threads::thread_priority::high);
        auto& hpstream3 =
            pool.get_next_stream(pika::threads::thread_priority::high);
        auto& hpstream4 =
            pool.get_next_stream(pika::threads::thread_priority::high);

        PIKA_TEST_EQ(hpstream1, hpstream3);
        PIKA_TEST_EQ(hpstream2, hpstream4);
        PIKA_TEST_NEQ(hpstream1, hpstream2);
    }

    {
        // A pool is reference counted
        cu::cuda_pool pool{};
        PIKA_TEST(pool.valid());
        PIKA_TEST(bool(pool));

        cu::cuda_pool pool2{pool};
        PIKA_TEST(pool2.valid());
        PIKA_TEST(bool(pool2));
        PIKA_TEST_EQ(pool, pool2);

        cu::cuda_pool pool3 = pool;
        PIKA_TEST(pool3.valid());
        PIKA_TEST(bool(pool3));
        PIKA_TEST_EQ(pool, pool3);

        cu::cuda_pool pool4{std::move(pool)};
        PIKA_TEST(!pool.valid());
        PIKA_TEST(!bool(pool));
        PIKA_TEST(pool4.valid());
        PIKA_TEST(bool(pool4));
        PIKA_TEST_NEQ(pool, pool4);

        cu::cuda_pool pool5{std::move(pool4)};
        PIKA_TEST(!pool4.valid());
        PIKA_TEST(!bool(pool4));
        PIKA_TEST(pool5.valid());
        PIKA_TEST(bool(pool5));
        PIKA_TEST_NEQ(pool4, pool5);
    }

    {
        // A pool can be used to schedule work
        int const n = 1000;
        int* p;
        cu::check_cuda_error(hipMalloc(&p, sizeof(int) * n));

        cu::cuda_pool pool{};

        for (std::size_t i = 0; i < n; ++i)
        {
            kernel<<<1, 1, 0, pool.get_next_stream().get()>>>(p, i);
            cu::check_cuda_error(hipGetLastError());
        }

        cu::check_cuda_error(hipDeviceSynchronize());
        std::vector<int> s(n, 0);

        cu::check_cuda_error(
            hipMemcpy(s.data(), p, sizeof(int) * n, hipMemcpyDeviceToHost));
        cu::check_cuda_error(hipFree(p));

        for (int i = 0; i < n; ++i)
        {
            PIKA_TEST_EQ(s[i], i * 2);
        }
    }
}
